#include "hip/hip_runtime.h"
// Copyright (c) 2021 CINN Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <glog/logging.h>
#include <gtest/gtest.h>

#include <random>
#include <vector>

#include "cinn/common/cuda_test_helper.h"
#include "cinn/common/float16.h"

namespace cinn {
namespace common {

__global__ void cast_fp32_to_fp16_cuda_kernel(const float* input, const int num, float16* out) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num) {
    out[idx] = float16(input[idx]);
  }
}

__global__ void cast_fp16_to_fp32_cuda_kernel(const float16* input, const int num, float* out) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num) {
    out[idx] = float(input[idx]);
  }
}

__global__ void test_fp16_cuda_kernel(const float16* x, const float16* y, const int num, float16* out) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num) {
    float16 x_i = x[idx], y_i = y[idx];
    x_i += float16(1);

    out[idx] = (x_i + y_i) * (x_i - y_i);
  }
}

__global__ void test_fp32_cuda_kernel(const float* x, const float* y, const int num, float* out) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num) {
    float x_i = x[idx], y_i = y[idx];
    x_i += 1.0f;

    out[idx] = (x_i + y_i) * (x_i - y_i);
  }
}

TEST(FP16, basic_cuda) {
#ifdef CUDA_VERSION
  LOG(INFO) << "CUDA version: " << CUDA_VERSION;
#endif

  int num = 2048;

  hipStream_t stream;
  CUDA_CALL(hipStreamCreate(&stream));

  dim3 block = 1024;
  dim3 grid  = (num + block.x - 1) / block.x;

  std::vector<float> x_fp32_host(num), y_fp32_host(num);
  {  // step1 : generate input data
    std::random_device r;
    std::default_random_engine eng(r());
    std::uniform_real_distribution<float> dis(1e-5f, 1.0f);

    for (int i = 0; i < num; ++i) {
      x_fp32_host[i] = dis(eng);
      y_fp32_host[i] = dis(eng);
    }
  }

  CudaMem x_fp32_device, y_fp32_device, out_fp32_device;
  {  // step2 : compute fp32 result
    auto x_fp32_ptr   = x_fp32_device.mutable_data<float>(num);
    auto y_fp32_ptr   = y_fp32_device.mutable_data<float>(num);
    auto out_fp32_ptr = out_fp32_device.mutable_data<float>(num);

    x_fp32_device.MemcpyFromHost(x_fp32_host.data(), num * sizeof(float), stream);
    y_fp32_device.MemcpyFromHost(y_fp32_host.data(), num * sizeof(float), stream);

    test_fp32_cuda_kernel<<<grid, block, 0, stream>>>(x_fp32_ptr, y_fp32_ptr, num, out_fp32_ptr);
  }

  CudaMem x_fp16_device, y_fp16_device, out_fp16_device;
  {  // step2 : compute fp16 result
    auto x_fp16_ptr   = x_fp16_device.mutable_data<float16>(num);
    auto y_fp16_ptr   = y_fp16_device.mutable_data<float16>(num);
    auto out_fp16_ptr = out_fp16_device.mutable_data<float16>(num);

    cast_fp32_to_fp16_cuda_kernel<<<grid, block, 0, stream>>>(x_fp32_device.data<float>(), num, x_fp16_ptr);
    cast_fp32_to_fp16_cuda_kernel<<<grid, block, 0, stream>>>(y_fp32_device.data<float>(), num, y_fp16_ptr);

    test_fp16_cuda_kernel<<<grid, block, 0, stream>>>(x_fp16_ptr, y_fp16_ptr, num, out_fp16_ptr);
  }

  CudaMem fp32res_fp16_device;
  {  // step3 : cast fp16 result to fp32 result
    auto fp32res_fp16_ptr = fp32res_fp16_device.mutable_data<float>(num);
    cast_fp16_to_fp32_cuda_kernel<<<grid, block, 0, stream>>>(out_fp16_device.data<float16>(), num, fp32res_fp16_ptr);
  }

  std::vector<float> out_fp32_host(num), out_fp16_host(num);
  {  // step4 : copy result from device to host
    out_fp32_device.MemcpyToHost(out_fp32_host.data(), num * sizeof(float), stream);
    fp32res_fp16_device.MemcpyToHost(out_fp16_host.data(), num * sizeof(float), stream);
  }

  hipStreamSynchronize(stream);

  for (int i = 0; i < num; ++i) {
    ASSERT_NEAR(out_fp32_host[i], out_fp16_host[i], 1e-2f);
  }

  hipStreamDestroy(stream);
}

}  // namespace common
}  // namespace cinn
